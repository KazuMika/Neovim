#include "hip/hip_runtime.h"
/*
 * FileName:     {{_file_name_}}
 * Author:       {{_author_}}
 * CreatedDate:  {{_date_}}
 * LastModified: 2023-02-26 13:30:39 +0900
 * Reference:    8ucchiman.jp
 * Description:  ---
 */


#include <stdio.h>
#define MACRO
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s: %d)\n", \
                    msg, hipGetErrorString(__err), \
                    __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

__global__ void hoge(void *arguments) {

}


#ifdef MACRO
int main(int argc, char* argv[]){
    {{_cursor_}}
    hoge<<numBlocks, threadsperblock>>>((void*) hoge);
    return 0;
}
#endif
